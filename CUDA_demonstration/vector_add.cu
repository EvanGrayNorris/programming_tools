
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

void hostVecAdd(float* A, float* B, float* C, int N){
      for(int i = 0; i < N; i = i + 1) {
          C[i] = A[i] + B[i];
      }
  }

int main()
{
  int N = 10000;
  size_t size = N * sizeof(float);

  //initialize host vectors
  float* host_A = (float*)malloc(size);
  float* host_B = (float*)malloc(size);
  float* host_C = (float*)malloc(size);

  //define host vectors
  for(int i = 0; i < N; i = i + 1) {
    host_A[i] = i;
    host_B[i] = i;
    //printf("A[%d] = %f, ", i,host_A[i]);
    //printf("B[%d] = %f \n", i,host_B[i]);
  
  }

  //initialize device vectors
  float* device_A;
  hipMalloc(&device_A,size);
  float* device_B;
  hipMalloc(&device_B,size);
  float* device_C;
  hipMalloc(&device_C,size);

  //copy host vector to device vector
  hipMemcpy(device_A, host_A, size, hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, size, hipMemcpyHostToDevice);

  //run device kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock -1) / threadsPerBlock;\
  //device vector adder
  //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(device_A, device_B, device_C, N);
  //host vector adder for comparison
  //hostVecAdd(host_A, host_B, host_C, N);

  //copy results from device to host
  hipMemcpy(host_C, device_C, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i = i + 1) {
     // printf("C[%d] = %f \n", i,host_C[i]);
  }

  //free device memory
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);
}