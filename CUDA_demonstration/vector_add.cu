
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

__global__ void MatAdd(float* A, float* B, float* C, int N)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x + blockDim.x + threadIdx.x;
    C[row*N+col] = A[row*N+col] + B[row*N+col];
}

__global__ void convert2Dto1D(float* X, float* Y, int m, int n){
  for(int i = 0; i < n; i = i + 1){
    for(int j = 0; j < m; j = j + 1){
      Y[j*n + i] = 1;
    }
  }

}

void hostVecAdd(float* A, float* B, float* C, int N){
      for(int i = 0; i < N; i = i + 1) {
          C[i] = A[i] + B[i];
      }
  }

int main()
{
  int N = 10000000000;
  size_t size = N * sizeof(float);

  //initialize host vectors
  float* host_A = (float*)malloc(size);
  float* host_B = (float*)malloc(size);
  float* host_C = (float*)malloc(size);

  //define host vectors
  for(int i = 0; i < N; i = i + 1) {
    if (i % 2)
    {
      host_A[i] = 1;
      host_B[i] = 0;
    }
    else{
      host_A[i] = 0;
      host_B[i] = 1;
    }
    //printf("A[%d] = %f, ", i,host_A[i]);
    //printf("B[%d] = %f \n", i,host_B[i]);
  
  }

  //initialize device vectors
  float* device_A;
  hipMalloc(&device_A,size);
  float* device_B;
  hipMalloc(&device_B,size);
  float* device_C;
  hipMalloc(&device_C,size);

  //copy host vector to device vector
  hipMemcpy(device_A, host_A, size, hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, size, hipMemcpyHostToDevice);

  //run device kernel
  int threadsPerBlock = 1024;
  int blocksPerGrid = (N + threadsPerBlock -1) / threadsPerBlock;
  //device vector adder
  MatAdd<<<blocksPerGrid, threadsPerBlock>>>(device_A, device_B, device_C, N);
  //host vector adder for comparison
  //hostVecAdd(host_A, host_B, host_C, N);

  //copy results from device to host
  hipMemcpy(host_C, device_C, size, hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i = i + 1) {
     //printf("C[%d] = %f \n", i,host_C[i]);
  }

  //free device memory
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);
}